#include "hip/hip_runtime.h"
/*
 * main.cpp
 *
 *  Created on: Sep 11, 2014
 *      Author: Abuenameh
 */

#include <ctime>

#include "cusimann.cuh"
#include "nelderMead.h"
#include "gutzwiller.hpp"

double f_nelderMead(unsigned int n, const double *x, double *grad,
		void *f_data) {
	return Energy<double>()(x, n, f_data);
}

int main(int argc, char** argv) {
	time_t start = time(NULL);

	real T_0 = 1000, T_min = 0.1;
	const unsigned int n = 2 * L * dim, N = 10;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*) malloc(sizeFD),
			f_cusimann_minimum;
	lb = (real*) malloc(sizeFD);
	unsigned int i;
	for (i = 0; i < n; i++)
		lb[i] = -1;
	ub = (real*) malloc(sizeFD);
	for (i = 0; i < n; i++)
		ub[i] = 1;

	unsigned int n_threads_per_block = 128;//512;//256;
	unsigned int n_blocks = 64;

	real U[L], J[L];
	for (int i = 0; i < L; i++) {
		U[i] = 1;
		J[i] = 0.001;
	}
	parameters<real> parms;
	parms.U = U;
	parms.J = J;
	parms.mu = 0.5;

	parameters<real>* d_parms;
	real* d_U;
	real* d_J;
	complex<real>* d_f;
	checkCudaErrors(hipMalloc(&d_U, L*sizeof(real)));
	checkCudaErrors(hipMemcpy(d_U, U, L*sizeof(real), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&d_J, L*sizeof(real)));
	checkCudaErrors(hipMemcpy(d_J, J, L*sizeof(real), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&d_f, L*sizeof(complex<real>)));
	checkCudaErrors(hipMemcpy(d_J, J, L*sizeof(complex<real>), hipMemcpyHostToDevice));

	real theta = 0;

	parms.U = d_U;
	parms.J = d_J;
	parms.f = d_f;
	parms.mu = 0.5;
	parms.theta = theta;
	parms.costh = cos(theta);
	parms.sinth = sin(theta);
	parms.cos2th = cos(2*theta);
	parms.sin2th = sin(2*theta);
	checkCudaErrors(hipMalloc(&d_parms, sizeof(parameters<real>)));
	checkCudaErrors(
			hipMemcpy(d_parms, &parms, sizeof(parameters<real>),
					hipMemcpyHostToDevice));

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb,
			ub, Energy<real>(), d_parms, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for (i = 0; i < n; i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	parms.U = U;
	parms.J = J;
//	parms.f = new complex<double>[L*dim];

	double f_nelderMead_minimum;
	double *nelderMead_minimum = (double*) malloc(n * sizeof(double));
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, &parms,
			nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for (i = 0; i < n; i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	delete[] parms.f;

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);

	time_t end = time(NULL);

	printf("Runtime: %ld s\n", end-start);

	return 0;
}

