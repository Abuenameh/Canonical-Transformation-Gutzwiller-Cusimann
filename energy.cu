#include "hip/hip_runtime.h"
/*
 * energy.cu
 *
 *  Created on: Sep 18, 2014
 *      Author: Abuenameh
 */

#include <stdio.h>

#include "gutzwiller.hpp"
#include "cuda_complex.hpp"

#define h(i, n) complex<T>(x[2*(i*dim+n)], x[2*(i*dim+n)+1])
#define f(i, n) complex<T>(x[2*(i*dim+n)], x[2*(i*dim+n)+1])

template<class T>
__host__  __device__ T Energy<T>::operator ()(const T *x, unsigned int n,
	void *f_data) const {

	parameters<T>* parms = (parameters<T>*) f_data;
	T* U = parms->U;
	T* J = parms->J;
	T mu = parms->mu;
	T theta = parms->theta;
//	T costh = parms->costh;
//	T sinth = parms->sinth;

//	parameters* parms = (parameters*) f_data;
//	real* U = parms->U;
//	real* J = parms->J;
//	real mu = parms->mu;
//	real theta = parms->theta;
//	real costh = parms->costh;
//	real sinth = parms->sinth;

	int q = blockDim.x;

	complex<T> expth = exp(complex<T>(0, 1) * theta);
	complex<T> expmth = ~expth;
	complex<T> exp2th = expth * expth;
	complex<T> expm2th = ~exp2th;

	complex<T> E = 0;

//	const complex<T>* ff = reinterpret_cast<const complex<T>*>(x);
//	const complex<T>* ffi[L];
//	for(int i = 0; i < L; i++) {
//		ffi[i] = ff;//+i*dim;//&ff[i*dim];
//	}
//	const complex<T> * f[L];
	T norm2[L];
//	complex<T> f2[dim];
//	for (int n = 0; n <= nmax; n++) {
//		f2[n] = complex<T>(x[2*n],x[2*n+1]);
//	}
	for (int i = 0; i < L; i++) {
//		f[i] = reinterpret_cast<const complex<T>*>(&x[2 * i * dim]);
//		f[i] = f2;
		norm2[i] = 0;
		for (int n = 0; n <= nmax; n++) {
			norm2[i] += norm(h(i,n)); //norm(f[i][n]);
			}
		}

//	typedef typename complextype<T>::type complex_t;
//
//	complex_t expth = complextype<T>::make_complex(costh, sinth);
//	complex_t expmth = ~expth;
//	complex_t exp2th = expth * expth;
//	complex_t expm2th = ~exp2th;
//
//	complex_t Ec = complex_t::zero();

//	const complex_t * f[L];
//	T norm2[L];
//	for (int i = 0; i < L; i++) {
//		f[i] = reinterpret_cast<const complex_t*>(&x[2 * i * dim]);
//		norm2[i] = 0;
//		for (int n = 0; n <= nmax; n++) {
//			norm2[i] += norm(f[i][n]);
//		}
//	}

	for (int i = 0; i < L; i++) {

		int k1 = mod(i - 2);
		int j1 = mod(i - 1);
		int j2 = mod(i + 1);
		int k2 = mod(i + 2);

		complex<T> E0 = 0;
		complex<T> E1j1 = 0;
		complex<T> E1j2 = 0;
		complex<T> E2j1 = 0;
		complex<T> E2j2 = 0;
		complex<T> E3j1 = 0;
		complex<T> E3j2 = 0;
		complex<T> E4j1j2 = 0;
		complex<T> E4j1k1 = 0;
		complex<T> E4j2k2 = 0;
		complex<T> E5j1j2 = 0;
		complex<T> E5j1k1 = 0;
		complex<T> E5j2k2 = 0;

//		complex_t E0 = complex_t::zero();
//		complex_t E1j1 = complex_t::zero();
//		complex_t E1j2 = complex_t::zero();
//		complex_t E2j1 = complex_t::zero();
//		complex_t E2j2 = complex_t::zero();
//		complex_t E3j1 = complex_t::zero();
//		complex_t E3j2 = complex_t::zero();
//		complex_t E4j1j2 = complex_t::zero();
//		complex_t E4j1k1 = complex_t::zero();
//		complex_t E4j2k2 = complex_t::zero();
//		complex_t E5j1j2 = complex_t::zero();
//		complex_t E5j1k1 = complex_t::zero();
//		complex_t E5j2k2 = complex_t::zero();

		complex<T> Eg;
		for (int n = 0; n <= nmax; n++) {
			E0 += ((T) 0.5 * U[i] * n * (n - 1) - mu * n) * ~f(i,n)* f(i,n);

			if (n < nmax) {
				E1j1 += -J[j1] * expth * g<T>(n, n + 1) * ~f(i,n + 1) * ~f(j1,n)
				* f(i,n) * f(j1,n + 1);
				E1j2 += -J[i] * expmth * g<T>(n, n + 1) * ~f(i,n + 1) * ~f(j2,n)* f(i,n)
                        * f(j2,n + 1);
            }

            if (n > 0) {
                E2j1 += (T)0.5 * J[j1] * J[j1] * exp2th * g<T>(n, n) * g<T>(n - 1, n + 1)
                        * ~f(i,n + 1) * ~f(j1,n - 1) * f(i,n - 1) * f(j1,n + 1)
                        * (1 / eps<T>(U, i, j1, n, n) - 1 / eps<T>(U, i, j1, n - 1, n + 1));
                E2j2 += (T)0.5 * J[i] * J[i] * expm2th * g<T>(n, n) * g<T>(n - 1, n + 1)
                        * ~f(i,n + 1) * ~f(j2,n - 1) * f(i,n - 1) * f(j2,n + 1)
                        * (1 / eps<T>(U, i, j2, n, n) - 1 / eps<T>(U, i, j2, n - 1, n + 1));
            }

            for (int m = 1; m <= nmax; m++) {
                if (n != m - 1) {
                    E3j1 += (T)0.5 * (J[j1] * J[j1] / eps<T>(U, i, j1, n, m)) * g<T>(n, m)
                            * g<T>(m - 1, n + 1)
                            * (~f(i,n + 1) * ~f(j1,m - 1) * f(i,n + 1) * f(j1,m - 1)
                            - ~f(i,n) * ~f(j1,m) * f(i,n) * f(j1,m));
                    E3j2 += (T)0.5 * (J[i] * J[i] / eps<T>(U, i, j2, n, m)) * g<T>(n, m)
                            * g<T>(m - 1, n + 1)
                            * (~f(i,n + 1) * ~f(j2,m - 1) * f(i,n + 1) * f(j2,m - 1)
                            - ~f(i,n) * ~f(j2,m) * f(i,n) * f(j2,m));
                }
            }

            if (n > 0) {
                E4j1j2 += (T)0.5 * (J[j1] * J[i] / eps<T>(U, i, j1, n, n)) * g<T>(n, n)
                        * g<T>(n - 1, n + 1) * ~f(i,n + 1) * ~f(j1,n - 1) * ~f(j2,n)
                        * f(i,n - 1) * f(j1,n) * f(j2,n + 1);
                E4j1j2 += (T)0.5 * (J[i] * J[j1] / eps<T>(U, i, j2, n, n)) * g<T>(n, n)
                        * g<T>(n - 1, n + 1) * ~f(i,n + 1) * ~f(j2,n - 1) * ~f(j1,n)
                        * f(i,n - 1) * f(j2,n) * f(j1,n + 1);
                E4j1k1 += (T)0.5 * (J[j1] * J[k1] / eps<T>(U, i, j1, n, n)) * g<T>(n, n)
                        * g<T>(n - 1, n + 1) * ~f(i,n + 1) * ~f(j1,n - 1) * ~f(k1,n)
                        * f(i,n) * f(j1,n + 1) * f(k1,n - 1);
                E4j2k2 += (T)0.5 * (J[i] * J[j2] / eps<T>(U, i, j2, n, n)) * g<T>(n, n)
                        * g<T>(n - 1, n + 1) * ~f(i,n + 1) * ~f(j2,n - 1) * ~f(k2,n)
                        * f(i,n) * f(j2,n + 1) * f(k2,n - 1);
                E4j1j2 -= (T)0.5 * (J[j1] * J[i] / eps<T>(U, i, j1, n - 1, n + 1))
                        * g<T>(n, n) * g<T>(n - 1, n + 1) * ~f(i,n + 1) * ~f(j1,n)
                        * ~f(j2,n - 1) * f(i,n - 1) * f(j1,n + 1) * f(j2,n);
                E4j1j2 -= (T)0.5 * (J[i] * J[j1] / eps<T>(U, i, j2, n - 1, n + 1))
                        * g<T>(n, n) * g<T>(n - 1, n + 1) * ~f(i,n + 1) * ~f(j2,n)
                        * ~f(j1,n - 1) * f(i,n - 1) * f(j2,n + 1) * f(j1,n);
                E4j1k1 -= (T)0.5 * (J[j1] * J[k1] / eps<T>(U, i, j1, n - 1, n + 1))
                        * g<T>(n, n) * g<T>(n - 1, n + 1) * ~f(i,n) * ~f(j1,n - 1)
                        * ~f(k1,n + 1) * f(i,n - 1) * f(j1,n + 1) * f(k1,n);
                E4j2k2 -= (T)0.5 * (J[i] * J[j2] / eps<T>(U, i, j2, n - 1, n + 1))
                        * g<T>(n, n) * g<T>(n - 1, n + 1) * ~f(i,n) * ~f(j2,n - 1)
                        * ~f(k2,n + 1) * f(i,n - 1) * f(j2,n + 1) * f(k2,n);
            }

            for (int m = 1; m <= nmax; m++) {
                if (n != m - 1 && n < nmax) {
                    E5j1j2 += (T)0.5 * (J[j1] * J[i] * exp2th / eps<T>(U, i, j1, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n + 1) * ~f(j1,m - 1)
                            * ~f(j2,m) * f(i,n + 1) * f(j1,m) * f(j2,m - 1);
                    E5j1j2 += (T)0.5 * (J[i] * J[j1] * expm2th / eps<T>(U, i, j2, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n + 1) * ~f(j2,m - 1)
                            * ~f(j1,m) * f(i,n + 1) * f(j2,m) * f(j1,m - 1);
                    E5j1k1 += (T)0.5 * (J[j1] * J[k1] * exp2th / eps<T>(U, i, j1, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n + 1) * ~f(j1,m - 1)
                            * ~f(k1,n) * f(i,n) * f(j1,m - 1) * f(k1,n + 1);
                    E5j2k2 += (T)0.5 * (J[i] * J[j2] * expm2th / eps<T>(U, i, j2, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n + 1) * ~f(j2,m - 1)
                            * ~f(k2,n) * f(i,n) * f(j2,m - 1) * f(k2,n + 1);
                    E5j1j2 -= (T)0.5 * (J[j1] * J[i] * exp2th / eps<T>(U, i, j1, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n) * ~f(j1,m - 1)
                            * ~f(j2,m) * f(i,n) * f(j1,m) * f(j2,m - 1);
                    E5j1j2 -= (T)0.5 * (J[i] * J[j1] * expm2th / eps<T>(U, i, j2, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n) * ~f(j2,m - 1)
                            * ~f(j1,m) * f(i,n) * f(j2,m) * f(j1,m - 1);
                    E5j1k1 -= (T)0.5 * (J[j1] * J[k1] * exp2th / eps<T>(U, i, j1, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n + 1) * ~f(j1,m)
                            * ~f(k1,n) * f(i,n) * f(j1,m) * f(k1,n + 1);
                    E5j2k2 -= (T)0.5 * (J[i] * J[j2] * expm2th / eps<T>(U, i, j2, n, m))
                            * g<T>(n, m) * g<T>(m - 1, n + 1) * ~f(i,n + 1) * ~f(j2,m)
                            * ~f(k2,n) * f(i,n) * f(j2,m) * f(k2,n + 1);
                }
            }
        }

		E += E0 / norm2[i];

		E += E1j1 / (norm2[i] * norm2[j1]);
		E += E1j2 / (norm2[i] * norm2[j2]);

		E += E2j1 / (norm2[i] * norm2[j1]);
		E += E2j2 / (norm2[i] * norm2[j2]);

		E += E3j1 / (norm2[i] * norm2[j1]);
		E += E3j2 / (norm2[i] * norm2[j2]);

		E += E4j1j2 / (norm2[i] * norm2[j1] * norm2[j2]);
		E += E4j1k1 / (norm2[i] * norm2[j1] * norm2[k1]);
		E += E4j2k2 / (norm2[i] * norm2[j2] * norm2[k2]);

		E += E5j1j2 / (norm2[i] * norm2[j1] * norm2[j2]);
		E += E5j1k1 / (norm2[i] * norm2[j1] * norm2[k1]);
		E += E5j2k2 / (norm2[i] * norm2[j2] * norm2[k2]);
	}

	return E.real();
}

template class Energy<float>
;
template class Energy<double>
;

