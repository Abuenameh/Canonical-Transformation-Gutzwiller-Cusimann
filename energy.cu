#include "hip/hip_runtime.h"
/*
 * energy.cu
 *
 *  Created on: Sep 18, 2014
 *      Author: Abuenameh
 */

#include "gutzwiller.hpp"

template<class T>
__host__ __device__ T Energy<T>::operator ()(const T *x, unsigned int n,
	void *f_data) const {
	parameters* parms = (parameters*) f_data;
	real* U = parms->U;
	real* J = parms->J;
	real mu = parms->mu;
	real costh = parms->costh;
	real sinth = parms->sinth;

	typedef typename complextype<T>::type complex_t;

	complex_t expth = complextype<T>::make_complex(costh, sinth);
	complex_t expmth = ~expth;
	complex_t exp2th = expth * expth;
	complex_t expm2th = ~exp2th;

	complex_t Ec = complex_t::zero();

	const complex_t * f[L];
	T norm2[L];
	for (int i = 0; i < L; i++) {
		f[i] = reinterpret_cast<const complex_t*>(&x[2 * i * dim]);
		norm2[i] = 0;
		for (int n = 0; n <= nmax; n++) {
			norm2[i] += norm(f[i][n]);
		}
	}

	for (int i = 0; i < L; i++) {

		int k1 = mod(i - 2);
		int j1 = mod(i - 1);
		int j2 = mod(i + 1);
		int k2 = mod(i + 2);

		complex_t E0 = complex_t::zero();
		complex_t E1j1 = complex_t::zero();
		complex_t E1j2 = complex_t::zero();
		complex_t E2j1 = complex_t::zero();
		complex_t E2j2 = complex_t::zero();
		complex_t E3j1 = complex_t::zero();
		complex_t E3j2 = complex_t::zero();
		complex_t E4j1j2 = complex_t::zero();
		complex_t E4j1k1 = complex_t::zero();
		complex_t E4j2k2 = complex_t::zero();
		complex_t E5j1j2 = complex_t::zero();
		complex_t E5j1k1 = complex_t::zero();
		complex_t E5j2k2 = complex_t::zero();

        for (int n = 0; n <= nmax; n++) {
            E0 += (0.5 * U[i] * n * (n - 1) - mu * n) * ~f[i][n] * f[i][n];

            if (n < nmax) {
                E1j1 += -J[j1] * expth * g(n, n + 1) * ~f[i][n + 1] * ~f[j1][n]
                        * f[i][n] * f[j1][n + 1];
                E1j2 += -J[i] * expmth * g(n, n + 1) * ~f[i][n + 1] * ~f[j2][n] * f[i][n]
                        * f[j2][n + 1];

            }

        }

		Ec += E0 / norm2[i];

		Ec += E1j1 / (norm2[i] * norm2[j1]);
		Ec += E1j2 / (norm2[i] * norm2[j2]);

		Ec += E2j1 / (norm2[i] * norm2[j1]);
		Ec += E2j2 / (norm2[i] * norm2[j2]);

		Ec += E3j1 / (norm2[i] * norm2[j1]);
		Ec += E3j2 / (norm2[i] * norm2[j2]);

		Ec += E4j1j2 / (norm2[i] * norm2[j1] * norm2[j2]);
		Ec += E4j1k1 / (norm2[i] * norm2[j1] * norm2[k1]);
		Ec += E4j2k2 / (norm2[i] * norm2[j2] * norm2[k2]);

		Ec += E5j1j2 / (norm2[i] * norm2[j1] * norm2[j2]);
		Ec += E5j1k1 / (norm2[i] * norm2[j1] * norm2[k1]);
		Ec += E5j2k2 / (norm2[i] * norm2[j2] * norm2[k2]);
	}

	return Ec.real();
}

template class Energy<float> ;
template class Energy<double> ;

