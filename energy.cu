#include "hip/hip_runtime.h"
/*
 * energy.cu
 *
 *  Created on: Sep 18, 2014
 *      Author: Abuenameh
 */

#include <stdio.h>

#include "gutzwiller.hpp"
#include "cuda_complex.hpp"

template<class T>
__host__ __device__ T Energy<T>::operator ()(const T *x, unsigned int n,
	void *f_data) const {

	parameters<T>* parms = (parameters<T>*) f_data;
	T* U = parms->U;
	T* J = parms->J;
	T mu = parms->mu;
	T theta = parms->theta;
//	T costh = parms->costh;
//	T sinth = parms->sinth;

//	parameters* parms = (parameters*) f_data;
//	real* U = parms->U;
//	real* J = parms->J;
//	real mu = parms->mu;
//	real theta = parms->theta;
//	real costh = parms->costh;
//	real sinth = parms->sinth;

	complex<T> expth = exp(complex<T>(0, 1) * theta);
	complex<T> expmth = ~expth;
	complex<T> exp2th = expth * expth;
	complex<T> expm2th = ~exp2th;

	complex<T> E = 0;
	T Er = 0;
	T Eri[L];

	const complex<T> * f[L];
	T norm2[L];
	complex<T> f2[dim];
	for (int n = 0; n <= nmax; n++) {
		f2[n] = complex<T>(x[2*n],x[2*n+1]);
	}
	for (int i = 0; i < L; i++) {
		f[i] = reinterpret_cast<const complex<T>*>(&x[2 * i * dim]);
//		f[i] = f2;
		norm2[i] = 0;
		for (int n = 0; n <= nmax; n++) {
			norm2[i] += norm(f[i][n]);
		}
	}

//	typedef typename complextype<T>::type complex_t;
//
//	complex_t expth = complextype<T>::make_complex(costh, sinth);
//	complex_t expmth = ~expth;
//	complex_t exp2th = expth * expth;
//	complex_t expm2th = ~exp2th;
//
//	complex_t Ec = complex_t::zero();

//	const complex_t * f[L];
//	T norm2[L];
//	for (int i = 0; i < L; i++) {
//		f[i] = reinterpret_cast<const complex_t*>(&x[2 * i * dim]);
//		norm2[i] = 0;
//		for (int n = 0; n <= nmax; n++) {
//			norm2[i] += norm(f[i][n]);
//		}
//	}

	for (int i = 0; i < L; i++) {

		int k1 = mod(i - 2);
		int j1 = mod(i - 1);
		int j2 = mod(i + 1);
		int k2 = mod(i + 2);

		complex<T> E0 = 0;
		complex<T> E1j1 = 0;
		complex<T> E1j2 = 0;
		complex<T> E2j1 = 0;
		complex<T> E2j2 = 0;
		complex<T> E3j1 = 0;
		complex<T> E3j2 = 0;
		complex<T> E4j1j2 = 0;
		complex<T> E4j1k1 = 0;
		complex<T> E4j2k2 = 0;
		complex<T> E5j1j2 = 0;
		complex<T> E5j1k1 = 0;
		complex<T> E5j2k2 = 0;

//		complex_t E0 = complex_t::zero();
//		complex_t E1j1 = complex_t::zero();
//		complex_t E1j2 = complex_t::zero();
//		complex_t E2j1 = complex_t::zero();
//		complex_t E2j2 = complex_t::zero();
//		complex_t E3j1 = complex_t::zero();
//		complex_t E3j2 = complex_t::zero();
//		complex_t E4j1j2 = complex_t::zero();
//		complex_t E4j1k1 = complex_t::zero();
//		complex_t E4j2k2 = complex_t::zero();
//		complex_t E5j1j2 = complex_t::zero();
//		complex_t E5j1k1 = complex_t::zero();
//		complex_t E5j2k2 = complex_t::zero();

		complex<T> Eg;
        for (int n = 0; n <= nmax; n++) {
////        	E0 += f[i][n];
//        	            complex<T> Ee = /*((T)0.5 * U[i] * n * (n - 1) - mu * n) */ ~f[i][n] * ~f[i][n];
//        	            complex<T> Ef = Ee;///*((T)0.5 * U[i] * n * (n - 1) - mu * n) */ ~f[i][n] * ~f[i][n];
//        	            Eg += Ee;
//        	printf("%d %d\n", i, n);
//        	complex<T> asd = f[i][n];
//        	const complex<T>* poi = f[0];
//        	const complex<T>* lkj = f[i];
//        	complex<T> oiu = poi[n];
//        	complex<T> kjh = lkj[n];
//        	complex<T> mnb = f[i][n];
//        	complex<T> zxc = f[L-1][nmax];
//        	complex<T> sdf = ~asd*asd;
//        	complex<T> wer = ~f[i][n]*f[i][n];
//            E0 = ((T)0.5 * U[i] * n * (n - 1) - mu * n) * ~f[i][n] * f[i][n];
//            E0 += ((T)0.5 * U[i] * n * (n - 1) - mu * n) * ~f2[n] * f2[n];
            E0 = ((T)0.5 * U[i] * n * (n - 1) - mu * n) * ~f[i][n] * f[i][n];

            if (n < nmax) {
//            	complex_t qwe = -J[j1]*expth*g(n,n+1)*f[i][n+1];//*~f[j1][n]*f[i][n]*f[j1][n+1];
//            	E1j1 += qwe;
//            	E1j1 += -J[j1]*expth*g(n,n+1)*f[i][n+1];
//            	E1j1 += -J[j1]*expth*g(n,n+1);
//            	E1j1 += expth*f[i][n+1];
//                E1j1 += -J[j1] * expth * g<T>(n, n + 1) * ~f[i][n + 1] * ~f[j1][n]
//                        * f[i][n] * f[j1][n + 1];
//                E1j2 += -J[i] * expmth * g<T>(n, n + 1) * ~f[i][n + 1] * ~f[j2][n] * f[i][n]
//                        * f[j2][n + 1];
            }

        }

		E += E0 / norm2[i];
//		Eri[i] = E0.real() / norm2[i] + E1j1.real() / (norm2[i] * norm2[j1]) + E1j2.real() / (norm2[i] * norm2[j2]);

		E += E1j1 / (norm2[i] * norm2[j1]);
		E += E1j2 / (norm2[i] * norm2[j2]);
//				Er += E1j1.real() / (norm2[i] * norm2[j1]);
//				Er += E1j2.real() / (norm2[i] * norm2[j2]);

//		E += E2j1 / (norm2[i] * norm2[j1]);
//		E += E2j2 / (norm2[i] * norm2[j2]);
//
//		E += E3j1 / (norm2[i] * norm2[j1]);
//		E += E3j2 / (norm2[i] * norm2[j2]);
//
//		E += E4j1j2 / (norm2[i] * norm2[j1] * norm2[j2]);
//		E += E4j1k1 / (norm2[i] * norm2[j1] * norm2[k1]);
//		E += E4j2k2 / (norm2[i] * norm2[j2] * norm2[k2]);
//
//		E += E5j1j2 / (norm2[i] * norm2[j1] * norm2[j2]);
//		E += E5j1k1 / (norm2[i] * norm2[j1] * norm2[k1]);
//		E += E5j2k2 / (norm2[i] * norm2[j2] * norm2[k2]);
	}
//    printf("Here\n");

	return E.real();
//	return Eri[0];
//	return 0;
}

template class Energy<float> ;
template class Energy<double> ;

