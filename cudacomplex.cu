#include "hip/hip_runtime.h"
/*
 * cudacomplex.cu
 *
 *  Created on: Sep 14, 2014
 *      Author: Abuenameh
 */

#include "cudacomplex.cuh"


//
// Non-member overloads for single complex
//

// subtract single complex from scalar
HOSTDEVICE _cudacomplex<float2, float> operator-(const float REF(a), const _cudacomplex<float2, float> REF(b)) {
    _cudacomplex<float2, float> result = {
        { a - b.value.x, -b.value.y}};
    return result;
}

// add single complex to scalar
HOSTDEVICE _cudacomplex<float2, float> operator+(const float REF(a), const _cudacomplex<float2, float> REF(b)) {
    _cudacomplex<float2, float> result = {
        { a + b.value.x, b.value.y}};
    return result;
}

// multiply scalar with single complex
HOSTDEVICE _cudacomplex<float2, float> operator*(const float REF(a), const _cudacomplex<float2, float> REF(b)) {
    _cudacomplex<float2, float> result = {
        { a * b.value.x, a * b.value.y}};
    return result;
}

// divide scalar by single complex
HOSTDEVICE _cudacomplex<float2, float> operator/(const float REF(a), const _cudacomplex<float2, float> REF(b)) {
    float tmp = (b.value.x * b.value.x + b.value.y * b.value.y);
    _cudacomplex<float2, float> result = {
        { (a * b.value.x) / tmp, (-a * b.value.y) / tmp}};
    return result;
}

//
// Non-member overloads for double complex
//

// subtract double complex from scalar
HOSTDEVICE _cudacomplex<double2, double> operator-(const double REF(a), const _cudacomplex<double2, double> REF(b)) {
    _cudacomplex<double2, double> result = {
        { a - b.value.x, -b.value.y}};
    return result;
}

// add double complex to scalar
HOSTDEVICE _cudacomplex<double2, double> operator+(const double REF(a), const _cudacomplex<double2, double> REF(b)) {
    _cudacomplex<double2, double> result = {
        { a + b.value.x, b.value.y}};
    return result;
}

// multiply scalar with double complex
HOSTDEVICE _cudacomplex<double2, double> operator*(const double REF(a), const _cudacomplex<double2, double> REF(b)) {
    _cudacomplex<double2, double> result = {
        { a * b.value.x, a * b.value.y}};
    return result;
}

// divide scalar by double complex
HOSTDEVICE _cudacomplex<double2, double> operator/(const double REF(a), const _cudacomplex<double2, double> REF(b)) {
    double tmp = (b.value.x * b.value.x + b.value.y * b.value.y);
    _cudacomplex<double2, double> result = {
        { (a * b.value.x) / tmp, (-a * b.value.y) / tmp}};
    return result;
}

// a possible alternative to a single complex constructor
HOSTDEVICE singlecomplex make_singlecomplex(float a, float b)
 {
    singlecomplex res;
    res.real() = a;
    res.imag() = b;
    return res;
}

// a possible alternative to a double complex constructor
HOSTDEVICE doublecomplex make_doublecomplex(double a, double b)
 {
    doublecomplex res;
    res.real() = a;
    res.imag() = b;
    return res;
}



